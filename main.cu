
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include <string.h>
#include <time.h>

#include <hiprand/hiprand_kernel.h>


//// macros for error checking
#ifndef NDEBUG
#define CUDA_CALL(x) do { \
cudaError_t err = x; \
if(err != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    printf("%s\n", hipGetErrorString(err)); \
}} while(0)
#define CURAND_CALL(x) do { \
cudaError_t err = x; \
if(err != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    printf("\t%s\n", hipGetErrorString(err)); \
}} while(0)

#else
#define CUDA_CALL(x) do { \
    x; \
}} while(0)
#define CURAND_CALL(x) do { \
    x; \
}} while(0)

#endif


//// parameters
struct metropolis_parameters
{
    double metropolis_offset;
    double xlower;
    double xupper;
    double x0;
    double xN;
    double a;
    size_t N;
    size_t N_until_equilibrium;
    size_t N_lattices;   
    size_t N_measure;
    size_t N_montecarlo;
    size_t N_markov;
    double Delta;

    double m0;
    double lambda;
    double mu_sq;
    double f_sq;
};

// __device__ double a;
// __device__ double f_sq;
// __device__ double m0 = 1.0;
// __device__ double mu_sq = 1.0;
// __device__ double lambda = 0.0;
// __device__ double epsilon = 1.;
// __device__ double Delta = 2.;

// double xlower = -2.;
// double xupper = 2.;
// size_t N = 2000;
// size_t max_kernels_per_block = 896;
const size_t max_threads_per_block = 512;


//// helper functions
__global__
void setup_randomize(hiprandState_t* state, size_t len)
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x; // TODO: is this correct?
    size_t stride = blockDim.x;

    for (unsigned int i=id; i<len; i+=stride) {
        hiprand_init(1234, i, 0, &state[i]);
    };
}

__global__
void randomize_double_array(double* array, size_t len, double lower, double upper, hiprandState_t* state)
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x; // TODO: is this correct?
    size_t stride = blockDim.x;
    hiprandState_t localState = state[id];

    for (unsigned int i=id; i<len; i+=stride) {
        array[i] = lower + (upper - lower) * hiprand_uniform_double(&localState);
    };

    state[id] = localState;
}

void printfl(double x)
{
    printf("%f\n", x);
}

void export_csv_double_1d(FILE* file, double* arr, size_t cols) // TODO: rename cols parameter
{
    for (int col=0; col<cols; col++) {
        fprintf(file, "%f%s", arr[col], (col==cols-1 ? "":","));
    };
    fprintf(file, "\n");
}

void export_csv_double_2d(FILE* file, double* arr, size_t pitch, size_t width, size_t height)
{
    for (int row=0; row<height; row++) {
        export_csv_double_1d(file, (double*)((char*)arr + row*pitch), width);
    };
}

size_t cuda_block_amount(size_t kernels, size_t max_kernels)
{
    return (int)ceil( (double)(kernels) / max_kernels );
}

//// big functions
__device__
double potential(double x, metropolis_parameters params)
{
    return 1./2. * pow(params.mu_sq, 2) * pow(x, 2) + params.lambda * pow(x, 4); // anharmonic oscillator potential
}

// __device__
// double potential_alt(double x)
// {
//     return lambda * pow( pow(x, 2.f) - f_sq, 2.f );
// }

__device__ double (*potential_ptr)(double, metropolis_parameters) = *potential;

__device__
double action_point(double x0, double x1, metropolis_parameters params)
{
    return params.a * (1./2. * params.m0 * pow((x1-x0), 2) / pow(params.a, 2) + (*potential_ptr)(x0, params));
}

__device__
double action_2p(double xm1, double x0, double x1, metropolis_parameters parameters)
{
    double action_0 = action_point(xm1, x0, parameters);
    double action_m1 = action_point(x0, x1, parameters);
    return action_0 + action_m1;
}

__global__ 
void action_latticeconf(double* lattice, metropolis_parameters params, double* action) 
{
    size_t idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= params.N) {
        return;
    };

    lattice += idx;
    *action += action_point(lattice[0], lattice[1], params);
}

__global__ 
void action_latticeconf_synchronous(double* lattice, metropolis_parameters params, double* action) 
{
    for (size_t i=0; i<params.N; i++) {
        *action += action_point(lattice[0], lattice[1], params);
        lattice += 1;
    };
}

void export_metropolis_data(const char filename[], double* ensemble, size_t pitch, size_t width, size_t height)
// write metropolis data. Takes in pointer to data on device memory
{
    double* ensemble_host;
    CUDA_CALL(hipHostMalloc(&ensemble_host, height * width*sizeof(double), hipHostMallocDefault));
    CUDA_CALL(hipMemcpy2D(ensemble_host, width*sizeof(double), ensemble, pitch, width*sizeof(double), height, hipMemcpyDeviceToHost));
    if (filename) {
        FILE* file = fopen(filename, "w");
        export_csv_double_2d(file, ensemble_host, width*sizeof(double), width, height);
        fclose(file);
    }

    CUDA_CALL(hipHostFree(ensemble_host));
}

__global__
void metropolis_step(double* xj, size_t n_points, size_t start_offset, metropolis_parameters params, hiprandState_t* random_state) 
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x;
    size_t offset = id * params.metropolis_offset + start_offset;
    if (offset >= n_points) { // do nothing if the point would be out of range
        return;
    } 
    xj = xj + offset;

    hiprandState_t localState = random_state[id];

    double xjp = hiprand_uniform_double(&localState) * (2*params.Delta) + *xj - params.Delta;
    double S_delta = action_2p(xj[-1], xjp, xj[1], params) - action_2p(xj[-1], *xj, xj[1], params);

    if (S_delta < 0) {
        *xj = xjp;
    }
    else {
        if (exp(-S_delta) > hiprand_uniform_double(&localState)) {
            *xj = xjp;
        };
    };

    random_state[id] = localState;
}

void metropolis_call(metropolis_parameters parameters, double* x, hiprandState* random_state, size_t metropolis_blocks, size_t metropolis_kernels) {
    for (size_t start_offset=0; start_offset<parameters.metropolis_offset; start_offset++) {
        for (size_t o=0; o<parameters.N_markov; o++) {
            metropolis_step
                <<<metropolis_blocks, metropolis_kernels>>>
                (x+1, parameters.N-1, start_offset, parameters, random_state);
            CUDA_CALL(hipDeviceSynchronize());
        };
    };
}

void metropolis_algo(metropolis_parameters parameters, double** ensemble_out, size_t* pitch, size_t* width, size_t* height)
// executes the metropolis algorithm, writes data into ensemble, pitch in bytes into pitch, width in doubles into width, height into height
{
    // parameters that are used directly
    size_t metropolis_offset = parameters.metropolis_offset; // offset between kernels. The smaller the number, the more kernels run in parallel. Minimum 2
    double xlower = parameters.xlower;
    double xupper = parameters.xupper;
    double x0 = parameters.x0;
    double xN = parameters.xN;
    size_t N = parameters.N;
    size_t N_until_equilibrium = parameters.N_until_equilibrium;
    size_t N_lattices = parameters.N_lattices;
    size_t N_measure = parameters.N_measure;
    size_t N_montecarlo = parameters.N_montecarlo;

    // determine kernel amounts
    size_t metropolis_kernels = (size_t)ceil( (double)(N-1) / metropolis_offset ); // amount of kernels that are run in parallel
    size_t metropolis_blocks = (size_t)ceil( (double)(metropolis_kernels) / max_threads_per_block );
    if (metropolis_blocks > 1) {
        metropolis_kernels = max_threads_per_block;
    }

    // initialize data arrays
    size_t N_measurements = N_lattices * N_measure;

    hiprandState_t *random_state, *random_state_algo;
    CUDA_CALL(hipMallocManaged(&random_state, (N-1) * sizeof(hiprandState_t)));
    CUDA_CALL(hipMallocManaged(&random_state_algo, (N-1) * sizeof(hiprandState_t)));
    setup_randomize<<<1, max_threads_per_block>>>(random_state, N-1); // NOTE: this could be parallelized more efficiently, but it probably doesn't make a significant difference
    setup_randomize<<<1, max_threads_per_block>>>(random_state_algo, metropolis_kernels); // NOTE: this could be parallelized more efficiently, but it probably doesn't make a significant difference
    hipDeviceSynchronize();
    
    double *x, *ensemble;
    CUDA_CALL(hipMallocManaged(&x, (N+1) * sizeof(double)));
    size_t ensemble_pitch;
    CUDA_CALL(hipMallocPitch(&ensemble, &ensemble_pitch, (N+1) * sizeof(double), N_measurements));

    x[0] = x0;
    x[N] = xN;
        
    // metropolis algorithm
    unsigned int measure_index = 0;
    for (int l=0; l<N_lattices; l++) {
        randomize_double_array<<<1, max_threads_per_block>>>(x+1, N-1, xlower, xupper, random_state);
        CUDA_CALL(hipDeviceSynchronize());

        // wait until equilibrium
        for (size_t j=0; j<N_until_equilibrium; j++) {
            metropolis_call(parameters, x, random_state_algo, metropolis_blocks, metropolis_kernels);
        }

        // start measuring
        for (size_t j=0; j<N_measure; j++) {
            for (size_t k=0; k<N_montecarlo; k++) {
                metropolis_call(parameters, x, random_state_algo, metropolis_blocks, metropolis_kernels);
            };
            // measure the new lattice configuration
            CUDA_CALL(hipMemcpy((double*)((char*)ensemble + ensemble_pitch*measure_index), x, (N+1)*sizeof(double), hipMemcpyDeviceToDevice));
            measure_index++;
        };
    };

    // return and cleanup
    *ensemble_out = ensemble;
    *pitch = ensemble_pitch;
    *width = N+1;
    *height = N_measurements;
    CUDA_CALL(hipFree(random_state));
    CUDA_CALL(hipFree(random_state_algo));
    CUDA_CALL(hipFree(x));
}



int main()
{
    // Query CUDA device properties
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %i\n", i);
        printf("Device name: %s\n", prop.name);
        printf("Max threads per block: %i\n", prop.maxThreadsPerBlock);
    }


    metropolis_parameters parameters = {
    .metropolis_offset = 2,
    .xlower = -2., .xupper = 2., .x0 = 0.0, .xN = 0.0,
    .a = 1., .N = 1000,
    .N_until_equilibrium = 100, .N_lattices = 3, .N_measure = 60, .N_montecarlo = 5, .N_markov = 1, .Delta = 2.0,
    .m0 = 1.0, .lambda = 0.0, .mu_sq = 1.0,
    .f_sq = -1.0 // placeholder value
    };

    double* ensemble;
    size_t pitch, width, height;
    metropolis_parameters params_4_5 = parameters;


    // step 1: plot action
    metropolis_parameters params_0 = parameters;
    params_0.m0 = .5;
    params_0.a = .5;
    params_0.N = 100;
    params_0.N_lattices = 1;
    params_0.N_until_equilibrium = 0;
    params_0.N_measure = 400;

    metropolis_algo(params_0, &ensemble, &pitch, &width, &height);

    double* actions; 
    CUDA_CALL((hipHostMalloc(&actions, height, hipHostMallocDefault)));

    // for (size_t i=0; i<height; i++) {
    //     printfl(actions[i]);
    // }

    size_t n_blocks = cuda_block_amount(params_0.N-1, max_threads_per_block);

    for (size_t i=0; i<height; i++) {
        // action_latticeconf<<<n_blocks, max_threads_per_block>>>((double*)((char*)ensemble + i*pitch), params_0, actions+i);
        action_latticeconf_synchronous<<<1, 1>>>((double*)((char*)ensemble + i*pitch), params_0, actions+i);
    };
    CUDA_CALL(hipDeviceSynchronize());

    FILE* file_action = fopen("action.csv", "w");
    export_csv_double_1d(file_action, actions, height);
    fclose(file_action);

    CUDA_CALL(hipFree(ensemble));
    CUDA_CALL(hipHostFree(actions));


    // step 2: Fig 4, 5
    metropolis_algo(params_4_5, &ensemble, &pitch, &width, &height);
    export_metropolis_data("harmonic_a.csv", ensemble, pitch, width, height);
    CUDA_CALL(hipFree(ensemble));

/*
    //// Fig. 6
    metropolis_parameters parameters_6 = parameters;
    parameters_6.N = 51;
    parameters_6.N_montecarlo = 20;
    parameters_6.mu_sq = 2.0;
    parameters_6.a = 0.5;
    parameters_6.Delta = 2 * sqrt(parameters.a);
    metropolis_algo(parameters_6, "harmonic_b.csv");
*/

    // TODO: use the f_sq potential from here on
    // potential_ptr = *potential_alt;

    // Fig. 7
    // metropolis_parameters parameters_7 = parameters;
    // parameters_7.N = 50;
    // parameters_7.N_lattices = 1;
    // parameters_7.N_measure = 1;
    // parameters_7.N_montecarlo = 40;
    // parameters_7.N_markov = 5;
    // parameters_7.lambda = 1.0;
    // parameters_7.a = 1.0;
    // parameters_7.Delta = 2 * sqrt(parameters.a);
    // parameters_7.m0 = 0.5;

    // parameters_7.f_sq = 0.5;
    // metropolis_algo(parameters_7, "anharmonic_a.csv");
    // parameters_7.f_sq = 1.0;
    // metropolis_algo(parameters_7, "anharmonic_b.csv");
    // parameters_7.f_sq = 2.0;
    // metropolis_algo(parameters_7, "anharmonic_c.csv");

    /*
    // Fig. 8
    m0 = 0.5;
    f_sq = 2.0;
    N = 200;
    epsilon = 0.25;
    metropolis_algo(0., 0., 10, 50, 10, 5, NULL, "anharmonic_e.csv");
    // metropolis_algo(0., 0., 100, 50, 10, 5, NULL, "anharmonic_d.csv");
    // metropolis_algo(0., 0., 100, 50, 1, 5, "anharmonic_d.csv", NULL);

    // //// Fig. 9
    m0 = 0.5;
    f_sq = 2.0;
    N = 303;
    a = 0.25;
    metropolis_algo(0., 0., 1, 10, 1, 5, NULL, "anharmonic_correlation_a.csv");
    metropolis_algo(0., 0., 1, 10, 1, 10, NULL, "anharmonic_correlation_b.csv");
    metropolis_algo(0., 0., 1, 10, 1, 15, NULL, "anharmonic_correlation_c.csv");
    */

} 


// TODO: fix end points (start and end should be regarded as the same point)