// clang -o a main.c -lm


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include <string.h>
#include <time.h>

#include <hiprand/hiprand_kernel.h>

//// parameters
// __device__ size_t N;
// __device__ double epsilon;
__device__ double a;
// __device__ double Delta;
// __device__ double m0;
// __device__ double mu_sq;
// __device__ double lambda;
__device__ double f_sq;
// __device__ double xlower = -2.;
// __device__ double xupper = 2.;
double xlower = -2.;
double xupper = 2.;


__device__ double m0 = 1.0;
__device__ double mu_sq = 1.0;
__device__ double lambda = 0.0;
int N = 1000;
__device__ double epsilon = 1.;
__device__ double Delta = 2.;




// // helper functions
// double frand(double lower, double upper) // TODO: do this on the graphics card
// {
//     // static int seed;
//     // seed = rand();
//     // srand(seed);
//     return lower + (upper - lower) * ((double)rand() / (double)RAND_MAX);
// }

__global__
void setup_randomize(hiprandState_t* state)
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x; // TODO: is this correct?
    hiprand_init(1234, id, 0, &state[id]);
}

__global__
void randomize_double_array(double* array, size_t len, double lower, double upper, hiprandState_t* state)
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x; // TODO: is this correct?
    hiprandState_t localState = state[id];

    size_t stride = blockDim.x;
    for (unsigned int i=0; i<len; i+=stride) {
        array[i] = lower + (upper - lower) * hiprand_uniform_double(state);
    };

    state[id] = localState;
}

void printfl(double x)
{
    printf("%f\n", x);
}

void export_csv_double_1d(FILE* file, double* arr, size_t cols) // TODO: rename cols parameter
{
    for (int col=0; col<cols; col++) {
        fprintf(file, "%f%s", arr[col], (col==cols-1 ? "":","));
    };
    fprintf(file, "\n");
}

void export_csv_double_2d(FILE* file, double* arr, size_t pitch, size_t width, size_t height)
{
    for (int row=0; row<height; row++) {
        export_csv_double_1d(file, (double*)((char*)arr + row*pitch), width);
    };
}


// big functions
__device__
double potential(double x)
{
    return 1./2. * pow(mu_sq, 2) * pow(x, 2) + lambda * pow(x, 4); // anharmonic oscillator potential
}

__device__
double potential_alt(double x)
{
    return lambda * pow( pow(x, 2.f) - f_sq, 2.f );
}

__device__ double (*potential_ptr)(double) = *potential;

__device__
double action_point(double x0, double x1)
{
    return epsilon * (1./2. * m0 * pow((x1-x0), 2) / pow(epsilon, 2) + (*potential_ptr)(x0));
}

__device__
double action_2p(double xm1, double x0, double x1)
{
    double action_0 = action_point(xm1, x0);
    double action_m1 = action_point(x0, x1);
    return action_0 + action_m1;
}

__global__
void metropolis_step(double* xj, hiprandState_t* random_state) 
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x; // TODO: is this correct?
    hiprandState_t localState = random_state[id];
    // double xjp = frand(xlower, xupper);
    double xjp = hiprand_uniform_double(&localState);
    // randomize_double_array(&xjp, 1, xlower, xupper, &localState);

    double S_delta = action_2p(xj[-1], xjp, xj[1]) - action_2p(xj[-1], *xj, xj[1]);

    if (S_delta < 0) {
        // if (fabs(*xj) < 0.15 && fabs(xj[-1]) < 0.15 && fabs(xj[1]) < 0.15) {
        // printf("%f, %f, %f | %f\n", xj[-1], *xj, xj[1], xjp);
        // printfl(S_delta);};
        *xj = xjp;
    }
    else {
        if (exp(-S_delta) > hiprand_uniform_double(&localState)) {
            // printf("a: %f %f\n", exp(-S_delta), test);
            *xj = xjp;
        };
    };

    random_state[id] = localState;
}


void metropolis_algo(
    double x0, double xN,
    size_t N_lattices, size_t N_measure, size_t N_montecarlo, size_t N_markov,
    const char filename[], const char equilibrium_filename[])
    // double ensemble[N_lattices*(1+N_measure)][N+1], double equilibrium_ensemble[N_lattices][N+1])
{
    // ensemble
    // const unsigned int N_measurements = N_lattices * (1 + N_measure); // NOTE: for including the initial random lattice configurations
    size_t N_measurements = N_lattices * N_measure;
    // double ensemble[N_measurements][N+1];
    // double equilibrium_ensemble[N_lattices][N+1]; // array containing the "finished" states;

    // double *ensemble, *equilibrium_ensemble;
    // size_t ensemble_pitch, equilibrium_pitch;
    // cudaMallocPitch(&ensemble, &ensemble_pitch, N+1, N_measurements);
    // cudaMallocPitch(&equilibrium_ensemble, &equilibrium_pitch, N+1, N_lattices);

    hiprandState_t* random_state;
    hipMallocManaged(&random_state, (N-1) * sizeof(double));
    setup_randomize<<<1, N-1>>>(random_state);
    
    double *x, *ensemble;
    hipMallocManaged(&x, (N+1) * sizeof(double));
    // cudaMallocPitch(&ensemble, &ensemble_pitch, (N+1)*sizeof(double), N_measurements);
    hipHostMalloc(&ensemble, (N+1) * N_measurements * sizeof(double), hipHostMallocDefault);
    size_t ensemble_pitch = (N+1)*sizeof(double);
    // double *ensemble = malloc(N_measurements * (N+1) * sizeof(double));

    printf("test2\n");

    // initialize boundary values
    x[0] = x0;
    x[N] = xN;

    // metropolis algorithm
    unsigned int measure_index = 0;
    for (int l=0; l<N_lattices; l++) {
        // use curand_uniform_double
        randomize_double_array<<<1, N-1>>>(x+1, N-1, xlower, xupper, random_state);

        for (int j=0; j<N_measure; j++) {
            for (int k=0; k<N_montecarlo; k++) {
                for (int i=1; i<N; i++) {
                    // N_markov metropolis steps on the lattice site 
                    for (int o=0; o<N_markov; o++) {
                        metropolis_step<<<1, 1>>>(x+i, random_state);
                    }
                };
            };
            // measure the new lattice configuration
            hipMemcpy((float*)((char*)ensemble + ensemble_pitch*measure_index), x, (N+1)*sizeof(double), hipMemcpyHostToHost);
            measure_index++;
        };
    }

    // write to csv
    if (filename) {
        FILE* file = fopen(filename, "w");
        export_csv_double_2d(file, ensemble, ensemble_pitch, N+1, N_measurements);
        fclose(file);
    }

    // if (equilibrium_filename) {
    //     FILE* equilibrium_file = fopen(equilibrium_filename, "w");
    //     export_csv_double_2d(equilibrium_file, N_measurements, N+1, ensemble);
    //     fclose(equilibrium_file);
    // // }
}

// double correlation_function(unsigned int rows, unsigned int cols, double ensemble[rows][cols], )



int main()
{
    srand(time(NULL));
    // srand(42);


    time_t time_start = time(NULL); // start measuring time

    // potential_ptr = *potential;

    //// Fig. 4, 5
    // m0 = 1.0;
    // mu_sq = 1.0;
    // lambda = 0.0;
    // N = 1000;
    // epsilon = 1.;
    // Delta = 2 * sqrt(epsilon);

    printf("test\n");
    metropolis_algo(0., 0., 3, 60, 5, 5, "harmonic_a.csv", NULL);

/*
    //// Fig. 6
    m0 = 0.5;
    mu_sq = 2.0;
    lambda = 0.0;
    N = 51;
    epsilon = 0.5;
    Delta = 2 * sqrt(epsilon);
    metropolis_algo(0., 0., 1, 10, 1, 5, "harmonic_b.csv", NULL);
    // metropolis_algo(0., 0., 50, 1, 50, 5, "harmonic_b.csv", NULL);
    // metropolis_algo(0., 0., 6, 60, 5, 5, NULL, "harmonic_b.csv");

    // use the f_sq potential from here on
    potential_ptr = *potential_alt;

    //// Fig. 7
    m0 = 0.5;
    lambda = 1.0;
    epsilon = 1.0;
    N = 50;

    f_sq = 0.5;
    metropolis_algo(0., 0., 1, 1, 40, 5, "anharmonic_a.csv", NULL);
    f_sq = 1.0;
    metropolis_algo(0., 0., 1, 1, 40, 5, "anharmonic_b.csv", NULL);
    f_sq = 2.0;
    metropolis_algo(0., 0., 1, 1, 40, 5, "anharmonic_c.csv", NULL);

    //// Fig. 8
    m0 = 0.5;
    f_sq = 2.0;
    N = 200;
    epsilon = 0.25;
    metropolis_algo(0., 0., 10, 50, 10, 5, NULL, "anharmonic_e.csv");
    // metropolis_algo(0., 0., 100, 50, 10, 5, NULL, "anharmonic_d.csv");
    // metropolis_algo(0., 0., 100, 50, 1, 5, "anharmonic_d.csv", NULL);

    // //// Fig. 9
    m0 = 0.5;
    f_sq = 2.0;
    N = 303;
    a = 0.25;
    metropolis_algo(0., 0., 1, 10, 1, 5, NULL, "anharmonic_correlation_a.csv");
    metropolis_algo(0., 0., 1, 10, 1, 10, NULL, "anharmonic_correlation_b.csv");
    metropolis_algo(0., 0., 1, 10, 1, 15, NULL, "anharmonic_correlation_c.csv");
    */

    time_t time_finish = time(NULL); // time measured until now


    const time_t total_time = difftime(time_finish, time_start);
    printf("total time taken: %fs\n", (double)total_time);




/*
    // bin the data
    double bin_lower = -5.;
    double bin_upper = 5.;
    const unsigned int N_bins = 30;

    double bins[N_bins];
    double bins_range[N_bins];

    bin_data(ensemble, N_lattices*(N+1), bins, N_bins, bin_lower, bin_upper);
    // bin_data(measurements, N_measurements*(N+1), bins, N_bins, bin_lower, bin_upper);
    bin_range(bins_range, N_bins, bin_lower, bin_upper);

    FILE* bin_file = fopen("bins.csv", "w");
    export_csv_double_1d(bin_file, N_bins, bins_range);
    export_csv_double_1d(bin_file, N_bins, bins);
    fclose(bin_file);

    printf("%i %i\n", N_measurements, measure_index);
    printf("%i %i\n", N_measurements, N+1);
*/
}