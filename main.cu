
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include <string.h>
#include <time.h>

#include <hiprand/hiprand_kernel.h>


//// macros for error checking
#ifndef NDEBUG
#define CUDA_CALL(x) do { \
cudaError_t err = x; \
if(err != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    printf("%s\n", hipGetErrorString(err)); \
}} while(0)
#define CURAND_CALL(x) do { \
cudaError_t err = x; \
if(err != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    printf("\t%s\n", hipGetErrorString(err)); \
}} while(0)

#else
#define CUDA_CALL(x) do { \
    x; \
}} while(0)
#define CURAND_CALL(x) do { \
    x; \
}} while(0)

#endif


//// parameters
struct metropolis_parameters
{
    double metropolis_offset;
    double xlower;
    double xupper;
    double x0;
    double xN;
    double a;
    size_t N;
    size_t N_until_equilibrium;
    size_t N_lattices;   
    size_t N_measure;
    size_t N_montecarlo;
    size_t N_markov;
    double Delta;

    double m0;
    double lambda;
    double mu_sq;
    double f_sq;
};

// __device__ double a;
// __device__ double f_sq;
// __device__ double m0 = 1.0;
// __device__ double mu_sq = 1.0;
// __device__ double lambda = 0.0;
// __device__ double epsilon = 1.;
// __device__ double Delta = 2.;

// double xlower = -2.;
// double xupper = 2.;
// size_t N = 2000;
// size_t max_kernels_per_block = 896;
const size_t max_threads_per_block = 512;


//// helper functions
__global__
void setup_randomize(hiprandState_t* state, size_t len)
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x; // TODO: is this correct?
    size_t stride = blockDim.x;

    for (unsigned int i=id; i<len; i+=stride) {
        hiprand_init(1234, id, 0, &state[i]);
    };
}

__global__
void randomize_double_array(double* array, size_t len, double lower, double upper, hiprandState_t* state)
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x; // TODO: is this correct?
    size_t stride = blockDim.x;
    hiprandState_t localState = state[id];

    for (unsigned int i=id; i<len; i+=stride) {
        array[i] = lower + (upper - lower) * hiprand_uniform_double(&localState);
    };

    state[id] = localState;
}

void printfl(double x)
{
    printf("%f\n", x);
}

void export_csv_double_1d(FILE* file, double* arr, size_t cols) // TODO: rename cols parameter
{
    for (int col=0; col<cols; col++) {
        fprintf(file, "%f%s", arr[col], (col==cols-1 ? "":","));
    };
    fprintf(file, "\n");
}

void export_csv_double_2d(FILE* file, double* arr, size_t pitch, size_t width, size_t height)
{
    for (int row=0; row<height; row++) {
        export_csv_double_1d(file, (double*)((char*)arr + row*pitch), width);
    };
}


//// big functions
__device__
double potential(double x, metropolis_parameters params)
{
    return 1./2. * pow(params.mu_sq, 2) * pow(x, 2) + params.lambda * pow(x, 4); // anharmonic oscillator potential
}

// __device__
// double potential_alt(double x)
// {
//     return lambda * pow( pow(x, 2.f) - f_sq, 2.f );
// }

__device__ double (*potential_ptr)(double, metropolis_parameters) = *potential;

__device__
double action_point(double x0, double x1, metropolis_parameters params)
{
    return params.a * (1./2. * params.m0 * pow((x1-x0), 2) / pow(params.a, 2) + (*potential_ptr)(x0, params));
}

__device__
double action_2p(double xm1, double x0, double x1, metropolis_parameters parameters)
{
    double action_0 = action_point(xm1, x0, parameters);
    double action_m1 = action_point(x0, x1, parameters);
    return action_0 + action_m1;
}

// __global__ double action(double* lattices, size_t n, double* actions)
// {
//     size_t idx = blockDim.x * blockIdx.x + threadIdx.x;

    
// }

__global__
void metropolis_step(double* xj, size_t n_points, size_t start_offset, metropolis_parameters parameters, hiprandState_t* random_state) 
{
    double Delta = parameters.Delta;

    size_t id = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState_t localState = random_state[id];

    // apply offset
    size_t offset = id * parameters.metropolis_offset + start_offset;
    if (offset >= n_points) { // do nothing if the point would be out of range
        // printf("Offset %i rejected \n", offset);
        return;
    } 
    xj = xj + offset;

    // double xjp = curand_uniform_double(&localState) * (xupper-xlower) + xlower;
    double xjp = hiprand_uniform_double(&localState) * (2*Delta) + *xj - Delta;
    double S_delta = action_2p(xj[-1], xjp, xj[1], parameters) - action_2p(xj[-1], *xj, xj[1], parameters);

    if (S_delta < 0) {
        *xj = xjp;
    }
    else {
        if (exp(-S_delta) > hiprand_uniform_double(&localState)) {
            *xj = xjp;
        };
    };

    random_state[id] = localState;
}


void metropolis_call(metropolis_parameters parameters, double* x, hiprandState* random_state, size_t metropolis_blocks, size_t metropolis_kernels) {
    for (size_t start_offset=0; start_offset<parameters.metropolis_offset; start_offset++) {
        for (size_t o=0; o<parameters.N_markov; o++) {
            metropolis_step
                <<<metropolis_blocks, metropolis_kernels>>>
                (x+1, parameters.N-1, start_offset, parameters, random_state);
            CUDA_CALL(hipDeviceSynchronize());
        };
    };
}

void metropolis_algo(metropolis_parameters parameters, const char filename[])
{
    // parameters that are used directly
    size_t metropolis_offset = parameters.metropolis_offset; // offset between kernels. The smaller the number, the more kernels run in parallel. Minimum 2
    double xlower = parameters.xlower;
    double xupper = parameters.xupper;
    double x0 = parameters.x0;
    double xN = parameters.xN;
    size_t N = parameters.N;
    size_t N_until_equilibrium = parameters.N_until_equilibrium;
    size_t N_lattices = parameters.N_lattices;
    size_t N_measure = parameters.N_measure;
    size_t N_montecarlo = parameters.N_montecarlo;

    // determine kernel amounts
    size_t metropolis_kernels = (int)ceil( (double)(N-1) / metropolis_offset ); // amount of kernels that are run in parallel
    size_t metropolis_blocks = (int)ceil( (double)(metropolis_kernels) / max_threads_per_block );
    if (metropolis_blocks > 1) {
        metropolis_kernels = max_threads_per_block;
    }

    // initialize data arrays
    size_t N_measurements = N_lattices * N_measure;

    hiprandState_t* random_state;
    CUDA_CALL(hipMallocManaged(&random_state, (N-1) * sizeof(hiprandState_t)));
    setup_randomize<<<1, max_threads_per_block>>>(random_state, N-1); // NOTE: this could be parallelized more efficiently, but it probably doesn't make a significant difference
    hipDeviceSynchronize();
    
    double *x, *ensemble;
    CUDA_CALL(hipMallocManaged(&x, (N+1) * sizeof(double)));
    CUDA_CALL(hipHostMalloc(&ensemble, (N+1) * N_measurements * sizeof(double), hipHostMallocDefault));
    size_t ensemble_pitch = (N+1)*sizeof(double);

    x[0] = x0;
    x[N] = xN;
        
    // metropolis algorithm
    unsigned int measure_index = 0;
    for (int l=0; l<N_lattices; l++) {
        randomize_double_array<<<1, max_threads_per_block>>>(x+1, N-1, xlower, xupper, random_state);
        CUDA_CALL(hipDeviceSynchronize());

        // wait until equilibrium
        for (size_t j=0; j<N_until_equilibrium; j++) {
            metropolis_call(parameters, x, random_state, metropolis_blocks, metropolis_kernels);
        }

        // start measuring
        for (size_t j=0; j<N_measure; j++) {
            for (size_t k=0; k<N_montecarlo; k++) {
                metropolis_call(parameters, x, random_state, metropolis_blocks, metropolis_kernels);
            };
            // measure the new lattice configuration
            CUDA_CALL(hipMemcpy((float*)((char*)ensemble + ensemble_pitch*measure_index), x, (N+1)*sizeof(double), hipMemcpyHostToHost));
            measure_index++;
        };
    };

    // write data and cleanup
    if (filename) {
        FILE* file = fopen(filename, "w");
        export_csv_double_2d(file, ensemble, ensemble_pitch, N+1, N_measurements);
        fclose(file);
    }
    CUDA_CALL(hipFree(random_state));
    CUDA_CALL(hipFree(x));
    CUDA_CALL(hipHostFree(ensemble));
}



int main()
{
    // Query CUDA device properties
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %i\n", i);
        printf("Device name: %s\n", prop.name);
        printf("Max threads per block: %i\n", prop.maxThreadsPerBlock);
    }


    metropolis_parameters parameters = {
    .metropolis_offset = 2,
    .xlower = -2., .xupper = 2., .x0 = 0.0, .xN = 0.0,
    .a = 1., .N = 1000,
    .N_until_equilibrium = 100, .N_lattices = 3, .N_measure = 60, .N_montecarlo = 5, .N_markov = 1, .Delta = 2.0,
    .m0 = 1.0, .lambda = 0.0, .mu_sq = 1.0,
    .f_sq = -1.0 // placeholder value
    };


    // Fig 4, 5
    metropolis_parameters parameters_4_5 = parameters;
    metropolis_algo(parameters_4_5, "harmonic_a.csv");

    //// Fig. 6
    metropolis_parameters parameters_6 = parameters;
    parameters_6.N = 51;
    parameters_6.N_montecarlo = 20;
    parameters_6.mu_sq = 2.0;
    parameters_6.a = 0.5;
    parameters_6.Delta = 2 * sqrt(parameters.a);
    metropolis_algo(parameters_6, "harmonic_b.csv");

    // TODO: use the f_sq potential from here on
    // potential_ptr = *potential_alt;

    // Fig. 7
    // metropolis_parameters parameters_7 = parameters;
    // parameters_7.N = 50;
    // parameters_7.N_lattices = 1;
    // parameters_7.N_measure = 1;
    // parameters_7.N_montecarlo = 40;
    // parameters_7.N_markov = 5;
    // parameters_7.lambda = 1.0;
    // parameters_7.a = 1.0;
    // parameters_7.Delta = 2 * sqrt(parameters.a);
    // parameters_7.m0 = 0.5;

    // parameters_7.f_sq = 0.5;
    // metropolis_algo(parameters_7, "anharmonic_a.csv");
    // parameters_7.f_sq = 1.0;
    // metropolis_algo(parameters_7, "anharmonic_b.csv");
    // parameters_7.f_sq = 2.0;
    // metropolis_algo(parameters_7, "anharmonic_c.csv");

    /*
    // Fig. 8
    m0 = 0.5;
    f_sq = 2.0;
    N = 200;
    epsilon = 0.25;
    metropolis_algo(0., 0., 10, 50, 10, 5, NULL, "anharmonic_e.csv");
    // metropolis_algo(0., 0., 100, 50, 10, 5, NULL, "anharmonic_d.csv");
    // metropolis_algo(0., 0., 100, 50, 1, 5, "anharmonic_d.csv", NULL);

    // //// Fig. 9
    m0 = 0.5;
    f_sq = 2.0;
    N = 303;
    a = 0.25;
    metropolis_algo(0., 0., 1, 10, 1, 5, NULL, "anharmonic_correlation_a.csv");
    metropolis_algo(0., 0., 1, 10, 1, 10, NULL, "anharmonic_correlation_b.csv");
    metropolis_algo(0., 0., 1, 10, 1, 15, NULL, "anharmonic_correlation_c.csv");
    */

} 


// TODO: fix end points (start and end should be regarded as the same point)