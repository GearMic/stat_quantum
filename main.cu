// clang -o a main.c -lm


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include <string.h>
#include <time.h>

#include <hiprand/hiprand_kernel.h>


// macros for error checking
#ifndef NDEBUG
#define CUDA_CALL(x) do { \
cudaError_t err = x; \
if(err != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    printf("%s\n", hipGetErrorString(err)); \
}} while(0)
#define CURAND_CALL(x) do { \
cudaError_t err = x; \
if(err != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    printf("%s\n", hipGetErrorString(err)); \
}} while(0)

#else
#define CUDA_CALL(x) do { \
    x; \
}} while(0)
#define CURAND_CALL(x) do { \
    x; \
}} while(0)

#endif


//// parameters
// __device__ size_t N;
// __device__ double epsilon;
__device__ double a;
// __device__ double Delta;
// __device__ double m0;
// __device__ double mu_sq;
// __device__ double lambda;
__device__ double f_sq;
// __device__ double xlower = -2.;
// __device__ double xupper = 2.;
double xlower = -2.;
double xupper = 2.;


__device__ double m0 = 1.0;
__device__ double mu_sq = 1.0;
__device__ double lambda = 0.0;
// size_t N = 257; // for testing
__device__ double epsilon = 1.;
__device__ double Delta = 2.;

size_t N = 2000;
const size_t max_threads_per_block = 512;



// // helper functions
__global__
void setup_randomize(hiprandState_t* state, size_t len)
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x; // TODO: is this correct?
    size_t stride = blockDim.x;

    for (unsigned int i=id; i<len; i+=stride) {
        hiprand_init(1245, id, 0, &state[i]);
    };
}

__global__
void randomize_double_array(double* array, size_t len, double lower, double upper, hiprandState_t* state)
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x; // TODO: is this correct?
    size_t stride = blockDim.x;
    hiprandState_t localState = state[id];

    for (unsigned int i=id; i<len; i+=stride) {
        array[i] = lower + (upper - lower) * hiprand_uniform_double(&localState);
    };

    state[id] = localState;
}

void printfl(double x)
{
    printf("%f\n", x);
}

void export_csv_double_1d(FILE* file, double* arr, size_t cols) // TODO: rename cols parameter
{
    for (int col=0; col<cols; col++) {
        fprintf(file, "%f%s", arr[col], (col==cols-1 ? "":","));
    };
    fprintf(file, "\n");
}

void export_csv_double_2d(FILE* file, double* arr, size_t pitch, size_t width, size_t height)
{
    for (int row=0; row<height; row++) {
        export_csv_double_1d(file, (double*)((char*)arr + row*pitch), width);
    };
}


// big functions
__device__
double potential(double x)
{
    return 1./2. * pow(mu_sq, 2) * pow(x, 2) + lambda * pow(x, 4); // anharmonic oscillator potential
}

__device__
double potential_alt(double x)
{
    return lambda * pow( pow(x, 2.f) - f_sq, 2.f );
}

__device__ double (*potential_ptr)(double) = *potential;

__device__
double action_point(double x0, double x1)
{
    return epsilon * (1./2. * m0 * pow((x1-x0), 2) / pow(epsilon, 2) + (*potential_ptr)(x0));
}

__device__
double action_2p(double xm1, double x0, double x1)
{
    double action_0 = action_point(xm1, x0);
    double action_m1 = action_point(x0, x1);
    return action_0 + action_m1;
}

__global__
void metropolis_step(double* xj, size_t n_points, size_t kernel_offset, size_t start_offset, double xlower, double xupper, hiprandState_t* random_state) 
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState_t localState = random_state[id];

    // apply offset
    size_t offset = id * kernel_offset + start_offset;
    if (offset >= n_points) { // do nothing if the point would be out of range
        // printf("Offset %i rejected \n", offset);
        return;
    } 
    xj = xj + offset;


    double xjp = hiprand_uniform_double(&localState) * (xupper-xlower) + xlower;

    double S_delta = action_2p(xj[-1], xjp, xj[1]) - action_2p(xj[-1], *xj, xj[1]);

    if (S_delta < 0) {
        *xj = xjp;
    }
    else {
        if (exp(-S_delta) > hiprand_uniform_double(&localState)) {
            *xj = xjp;
        };
    };

    random_state[id] = localState;
}


void metropolis_algo(
    double x0, double xN,
    size_t N_lattices, size_t N_measure, size_t N_montecarlo, size_t N_markov,
    const char filename[], const char equilibrium_filename[])
{
    // determine kernel amounts
    size_t metropolis_offset = 2; // offset between kernels. The smaller the number, the more kernels run in parallel. Minimum 2
    size_t metropolis_kernels = (int)ceil( (double)(N-1) / metropolis_offset ); // amount of kernels that are run in parallel

    // size_t max_kernels_per_block = 896;
    size_t metropolis_blocks = (int)ceil( (double)(metropolis_kernels) / max_threads_per_block );

    if (metropolis_blocks > 1) {
        metropolis_kernels = max_threads_per_block;
    }


    size_t N_measurements = N_lattices * N_measure;

    hiprandState_t* random_state;
    CUDA_CALL(hipMallocManaged(&random_state, (N-1) * sizeof(hiprandState_t)));
    setup_randomize<<<1, max_threads_per_block>>>(random_state, N-1); // NOTE: this could be parallelized more efficiently, but it probably doesn' make a significant difference
    hipDeviceSynchronize();
    
    double *x, *ensemble;
    CUDA_CALL(hipMallocManaged(&x, (N+1) * sizeof(double)));
    // cudaMallocPitch(&ensemble, &ensemble_pitch, (N+1)*sizeof(double), N_measurements);
    CUDA_CALL(hipHostMalloc(&ensemble, (N+1) * N_measurements * sizeof(double), hipHostMallocDefault));
    size_t ensemble_pitch = (N+1)*sizeof(double);

    // initialize boundary values
    x[0] = x0;
    x[N] = xN;
        
    // metropolis algorithm
    unsigned int measure_index = 0;
    for (int l=0; l<N_lattices; l++) {
        randomize_double_array<<<1, max_threads_per_block>>>(x+1, N-1, xlower, xupper, random_state);
        CUDA_CALL(hipDeviceSynchronize());

        for (size_t j=0; j<N_measure; j++) {
            for (size_t k=0; k<N_montecarlo; k++) {
                for (size_t start_offset=0; start_offset<metropolis_offset; start_offset++) {
                    for (size_t o=0; o<N_markov; o++) {
                        metropolis_step<<<metropolis_blocks, metropolis_kernels>>>(x+1, N-1, metropolis_offset, start_offset, xlower, xupper, random_state);
                        CUDA_CALL(hipDeviceSynchronize());
                    };
                };
            };
            // measure the new lattice configuration
            CUDA_CALL(hipMemcpy((float*)((char*)ensemble + ensemble_pitch*measure_index), x, (N+1)*sizeof(double), hipMemcpyHostToHost));
            measure_index++;
        };
    };

    // write to csv
    if (filename) {
        FILE* file = fopen(filename, "w");
        export_csv_double_2d(file, ensemble, ensemble_pitch, N+1, N_measurements);
        fclose(file);
    }

    // free
    CUDA_CALL(hipFree(random_state));
    CUDA_CALL(hipFree(x));
    // CUDA_CALL(cudaFree(ensemble));
    hipFree(ensemble);
}



int main()
{
    // Query CUDA device properties
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %i\n", i);
        printf("Device name: %s\n", prop.name);
        printf("Max threads per block: %i\n", prop.maxThreadsPerBlock);
        // printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
        // printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        // printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
    

    time_t time_start = time(NULL); // start measuring time

    // potential_ptr = *potential;

    //// Fig. 4, 5
    // m0 = 1.0;
    // mu_sq = 1.0;
    // lambda = 0.0;
    // N = 1000;
    // epsilon = 1.;
    // Delta = 2 * sqrt(epsilon);

    metropolis_algo(0., 0., 3, 5, 5, 1, "harmonic_a.csv", NULL);


    time_t time_finish = time(NULL); // time measured until now

    const time_t total_time = difftime(time_finish, time_start);
    printf("total time taken: %fs\n", (double)total_time);



/*
    //// Fig. 6
    m0 = 0.5;
    mu_sq = 2.0;
    lambda = 0.0;
    N = 51;
    epsilon = 0.5;
    Delta = 2 * sqrt(epsilon);
    metropolis_algo(0., 0., 1, 10, 1, 5, "harmonic_b.csv", NULL);
    // metropolis_algo(0., 0., 50, 1, 50, 5, "harmonic_b.csv", NULL);
    // metropolis_algo(0., 0., 6, 60, 5, 5, NULL, "harmonic_b.csv");

    // use the f_sq potential from here on
    potential_ptr = *potential_alt;

    //// Fig. 7
    m0 = 0.5;
    lambda = 1.0;
    epsilon = 1.0;
    N = 50;

    f_sq = 0.5;
    metropolis_algo(0., 0., 1, 1, 40, 5, "anharmonic_a.csv", NULL);
    f_sq = 1.0;
    metropolis_algo(0., 0., 1, 1, 40, 5, "anharmonic_b.csv", NULL);
    f_sq = 2.0;
    metropolis_algo(0., 0., 1, 1, 40, 5, "anharmonic_c.csv", NULL);

    //// Fig. 8
    m0 = 0.5;
    f_sq = 2.0;
    N = 200;
    epsilon = 0.25;
    metropolis_algo(0., 0., 10, 50, 10, 5, NULL, "anharmonic_e.csv");
    // metropolis_algo(0., 0., 100, 50, 10, 5, NULL, "anharmonic_d.csv");
    // metropolis_algo(0., 0., 100, 50, 1, 5, "anharmonic_d.csv", NULL);

    // //// Fig. 9
    m0 = 0.5;
    f_sq = 2.0;
    N = 303;
    a = 0.25;
    metropolis_algo(0., 0., 1, 10, 1, 5, NULL, "anharmonic_correlation_a.csv");
    metropolis_algo(0., 0., 1, 10, 1, 10, NULL, "anharmonic_correlation_b.csv");
    metropolis_algo(0., 0., 1, 10, 1, 15, NULL, "anharmonic_correlation_c.csv");
    */





/*
    // bin the data
    double bin_lower = -5.;
    double bin_upper = 5.;
    const unsigned int N_bins = 30;

    double bins[N_bins];
    double bins_range[N_bins];

    bin_data(ensemble, N_lattices*(N+1), bins, N_bins, bin_lower, bin_upper);
    // bin_data(measurements, N_measurements*(N+1), bins, N_bins, bin_lower, bin_upper);
    bin_range(bins_range, N_bins, bin_lower, bin_upper);

    FILE* bin_file = fopen("bins.csv", "w");
    export_csv_double_1d(bin_file, N_bins, bins_range);
    export_csv_double_1d(bin_file, N_bins, bins);
    fclose(bin_file);

    printf("%i %i\n", N_measurements, measure_index);
    printf("%i %i\n", N_measurements, N+1);
*/
}