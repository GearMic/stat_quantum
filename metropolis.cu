#include "hip/hip_runtime.h"
#include <metropolis.cuh>
#include <helper.cuh>

__device__ double potential(double x, metropolis_parameters params)
{
    if (params.alt_potential)
        return params.lambda * pow( (pow(x, 2.) - params.f_sq), 2. ); // alt potential
    else
        return 1./2. * params.mu_sq * pow(x, 2.) + params.lambda * pow(x, 4.);
}

__device__ double action_point(double x0, double x1, metropolis_parameters params)
{
    return params.a * (1./2. * params.m0 * pow((x1-x0), 2) / pow(params.a, 2) + potential(x0, params));
}

__device__ double action_2p(double xm1, double x0, double x1, metropolis_parameters parameters)
{
    double action_0 = action_point(xm1, x0, parameters);
    double action_m1 = action_point(x0, x1, parameters);
    return action_0 + action_m1;
}

__global__ void action_latticeconf_synchronous(double* lattice, metropolis_parameters params, double* action) 
{
    for (size_t i=0; i<params.N; i++) {
        *action += action_point(lattice[0], lattice[1], params);
        lattice += 1;
    };
}

void export_metropolis_data(const char filename[], double* ensemble, size_t pitch, size_t width, size_t height)
// write metropolis data. Takes in pointer to data on device memory
{
    double* ensemble_host;
    CUDA_CALL(hipHostMalloc(&ensemble_host, height * width*sizeof(double)));
    CUDA_CALL(hipMemcpy2D(ensemble_host, width*sizeof(double), ensemble, pitch, width*sizeof(double), height, hipMemcpyDeviceToHost));
    if (filename) {
        FILE* file = fopen(filename, "w");
        export_csv_double_2d(file, ensemble_host, width*sizeof(double), width, height);
        fclose(file);
    }

    CUDA_CALL(hipHostFree(ensemble_host));
}

__global__ void metropolis_step(double* xj, size_t start_offset, metropolis_parameters params, hiprandState_t* random_state) 
{ // TODO: try only making changes at the very end
    size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    size_t offset = start_offset + idx * params.metropolis_offset;
    if (offset >= params.N) { // do nothing if the point would be out of range
        return;
    } 

    hiprandState_t localState = random_state[offset];

    double xjp = hiprand_uniform_double(&localState) * (2*params.Delta) - params.Delta + xj[offset];
    double lneighbor = xj[offset - 1];
    double rneighbor = xj[(offset + 1) % params.N];
    double S_delta = action_2p(lneighbor, xjp, rneighbor, params) - action_2p(lneighbor, xj[offset], rneighbor, params);

    if (S_delta <= 0) {
        xj[offset] = xjp;
    }
    else {
        double r = hiprand_uniform_double(&localState);
        if (exp(-S_delta) > r) {
            xj[offset] = xjp;
        };
    };

    random_state[offset] = localState;
}

void metropolis_call(metropolis_parameters params, double* x, hiprandState* random_state, size_t metropolis_blocks, size_t metropolis_kernels) {
    for (size_t start_offset=0; start_offset<params.metropolis_offset; start_offset++) {
        for (size_t o=0; o<params.N_markov; o++) {
            metropolis_step
                <<<metropolis_blocks, metropolis_kernels>>>
                (x, start_offset, params, random_state);
            CUDA_CALL(hipDeviceSynchronize());
        };
    };
}

void metropolis_algo(metropolis_parameters params, double** ensemble_out, size_t* pitch, size_t* width, size_t* height)
// executes the metropolis algorithm, writes data into ensemble, pitch in bytes into pitch, width in doubles into width, height into height
{
    size_t metropolis_offset = params.metropolis_offset; // offset between kernels. The smaller the number, the more kernels run in parallel. Minimum 2
    size_t N = params.N;

    // determine kernel amounts
    size_t metropolis_kernels = (size_t)ceil( (double)N/metropolis_offset ); // amount of kernels that are run in parallel
    size_t metropolis_blocks = cuda_block_amount(metropolis_kernels, max_threads_per_block);
    size_t threads_per_block = metropolis_kernels;
    if (metropolis_blocks > 1) {
        threads_per_block = max_threads_per_block;
    }
    
    printf("total: %zd\tblocks: %zd\t perblock: %zd\n", metropolis_kernels, metropolis_blocks, threads_per_block);

    // initialize data arrays
    size_t N_measurements = params.N_measure;

    hiprandState_t *random_state;
    CUDA_CALL(hipMallocManaged(&random_state, N*sizeof(hiprandState_t)));
    setup_randomize<<<1, max_threads_per_block>>>(random_state, N, random_seed);
    hipDeviceSynchronize();
    
    double *x, *ensemble;
    CUDA_CALL(hipMallocManaged(&x, N*sizeof(double)));
    size_t ensemble_pitch;
    CUDA_CALL(hipMallocPitch(&ensemble, &ensemble_pitch, N*sizeof(double), N_measurements));

    // metropolis algorithm
    unsigned int measure_index = 0;
    randomize_double_array<<<1, max_threads_per_block>>>(x, N, params.xlower, params.xupper, random_state);
    CUDA_CALL(hipDeviceSynchronize());

    // wait until equilibrium
    for (size_t j=0; j<params.N_until_equilibrium; j++) {
        metropolis_call(params, x, random_state, metropolis_blocks, threads_per_block);
    }

    // start measuring
    for (size_t j=0; j<params.N_measure; j++) {
        for (size_t k=0; k<params.N_montecarlo; k++) {
            metropolis_call(params, x, random_state, metropolis_blocks, threads_per_block);
        };
        // measure the new lattice configuration
        CUDA_CALL(hipMemcpy((double*)((char*)ensemble + ensemble_pitch*measure_index), x, N*sizeof(double), hipMemcpyDeviceToDevice));
        measure_index++;
    };

    // cleanup and return
    CUDA_CALL(hipFree(random_state));
    CUDA_CALL(hipFree(x));
    *ensemble_out = ensemble;
    *pitch = ensemble_pitch;
    *width = N;
    *height = N_measurements;
}

void metropolis_allinone(metropolis_parameters params, const char* filename)
{
    double* ensemble;
    size_t pitch, width, height;
    metropolis_algo(params, &ensemble, &pitch, &width, &height);
    export_metropolis_data(filename, ensemble, pitch, width, height);
    CUDA_CALL(hipFree(ensemble));
}